#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <omp.h>
#include <iostream>
#include <complex.h>
#include <math.h>

#include "kd_tree.h"


//Error checking GPU calls
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Mode 0 is CPU implementation
// Mode 1 is GPU implementation
#define MODE 0

//Define any constants here
#define BLOCKSIZE 128

using namespace std;


//function prototypes
void warmUpGPU();
void checkParams(unsigned int N, unsigned int DIM);
void calcDistMatCPU(float* dataset, unsigned int N, unsigned int DIM);

int main(int argc, char* argv[])
{
    printf("\nMODE: %d", MODE);
    warmUpGPU();

    char inputFname[500];
    unsigned int N = 0;
    unsigned int DIM = 0;
    float epsilon = 0;


    if (argc != 5)
    {
        fprintf(stderr, "Please provide the following on the command line: N (number of lines in the file), dimensionality (number of coordinates per point), epsilon, dataset filename.\n");
        exit(0);
    }

    sscanf(argv[1],"%d",&N);
    sscanf(argv[2],"%d",&DIM);
    sscanf(argv[3],"%f",&epsilon);
    strcpy(inputFname,argv[4]);

    checkParams(N, DIM);

    printf(
        "\nAllocating the following amount of memory for the dataset: %f GiB",
        (sizeof(float) * N * DIM) / (1024 * 1024 * 1024.0)
    );
    printf(
        "\nAllocating the following amount of memory for the distance matrix: %f GiB",
        (sizeof(float) * N * N) / (1024 * 1024 * 1024.0)
    );


    float* dataset = (float*)malloc(sizeof(float*) * N * DIM);
    importDataset(inputFname, N, DIM, dataset);


    //CPU-only mode
    //It only computes the distance matrix but does not query the distance matrix
    if (MODE == 0)
    {
        // Calculate with CPU implementation
        // TODO: IMPLEMENT CPU IMPLEMENTATION
        calcDistMatCPU(dataset, N, DIM);

        return(0);
    }

    double tstart=omp_get_wtime();

    //Allocate memory for the dataset
    float* dev_dataset;
    gpuErrchk(hipMalloc((float**)&dev_dataset, sizeof(float) * N * DIM));
    gpuErrchk(hipMemcpy(dev_dataset, dataset, sizeof(float) * N * DIM, hipMemcpyHostToDevice));

    //For part 1 that computes the distance matrix
    float* dev_distanceMatrix;
    gpuErrchk(hipMalloc((float**)&dev_distanceMatrix, sizeof(float) * N * N));


    //For part 2 for querying the distance matrix
    unsigned int* resultSet = (unsigned int*)calloc(N, sizeof(unsigned int));
    unsigned int* dev_resultSet;
    gpuErrchk(hipMalloc((unsigned int**)&dev_resultSet, sizeof(unsigned int) * N));
    gpuErrchk(hipMemcpy(dev_resultSet, resultSet, sizeof(unsigned int) * N, hipMemcpyHostToDevice));


    //Baseline kernels
    if (MODE == 1)
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N*1.0/BLOCKDIM);

        // Call baseline kernel here
        // TODO: IMPLEMENT GPU IMPLEMENTATION
    }


    //Copy result set from the GPU
    gpuErrchk(hipMemcpy(resultSet, dev_resultSet, sizeof(unsigned int) * N, hipMemcpyDeviceToHost));

    //Compute the sum of the result set array
    unsigned int totalWithinEpsilon = 0;

    //Write code here
    for(int resultIndex = 0; resultIndex < N; resultIndex += 1)
    {
        totalWithinEpsilon += resultSet[resultIndex];
    }

    printf("\nTotal number of points within epsilon: %u", totalWithinEpsilon);

    double tend = omp_get_wtime();

    printf("\n[MODE: %d, N: %d] Total time: %f", MODE, N, tend-tstart);


    printf("\n\n");
    return 0;
}


void warmUpGPU()
{
    printf("\nWarming up GPU for time trialing...\n");
    hipDeviceSynchronize();
    return;
}


void checkParams(unsigned int N, unsigned int DIM)
{
    if (N <= 0 || DIM <= 0)
    {
        fprintf(stderr, "\n Invalid parameters: Error, N: %u, DIM: %u", N, DIM);
        fprintf(stderr, "\nReturning");
        exit(0);
    }
}


void calcDistMatCPU(float* dataset, unsigned int N, unsigned int DIM)
{
    // initialize kd-tree with first point

    // loop over points
    {
        // add point to tree
    }
}
