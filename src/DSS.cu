#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <omp.h>
#include <iostream>
#include <complex.h>
#include <math.h>

#include "../include/kd_tree.cuh"


//Error checking GPU calls
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Mode 0 is CPU brute force - for checking tree results
// Mode 1 is CPU sequential implementation of kd-tree
// Mode 2 brute for query with GPU - for checking and comparing
// Mode 3 uses CPU to build the kd-tree and move to GPU, then uses GPU to query
// Mode 4 uses shared memory to move nodes in question to the tree
// Mode 5 uses 2D block
// Mode 6 uses 3D block
// ...
// #define MODE 0

//Define any constants here
#define BLOCKSIZE 128

using namespace std;


//function prototypes
void warmUpGPU();
void checkParams(unsigned int N, unsigned int DIM);

// cpu code
// brute force
void calcDistMatCPU(
    double* distanceMatrix,
    double* dataset,
    const unsigned int N,
    const unsigned int DIM
);
void queryDistMat(
    unsigned int* result,
    double* distanceMatrix,
    const double epsilon,
    const unsigned int N
);

// kd-tree
kd_tree_cpu* buildKdTreeCPU(const double* dataset, const unsigned int N, const unsigned int DIM);
void queryKdTreeCPU(
    struct kd_tree_cpu** tree,
    unsigned int* result,
    const double* dataset,
    const double epsilon,
    const unsigned int N,
    const unsigned int DIM
);

// gpu code
//__global__ void queryKdTreeGPU(struct kd_tree_node_gpu* gpu_nodes_array, float* testData);
__global__ void init_node_data(struct kd_tree_node_gpu* gpu_nodes_array, float* data, int insert_index);

// brute force?
__global__ void calcDistMatGPU(
    double* distanceMatrix,
    double* dataset,
    const unsigned int N,
    const unsigned int DIM
);


__global__ void queryDistMatGPU(
    unsigned int* result,
    double* distanceMatrix,
    const double epsilon,
    const unsigned int N
);


// query kd-tree
__global__ void queryKdTreeGPU(
    struct kd_tree_gpu** tree,
    unsigned int* result,
    double* dataset,
    const double epsilon,
    const unsigned int N,
    const unsigned int DIM
);


__global__ void queryKdTreeGPUWithSharedMem(
    struct kd_tree_gpu** tree,
    unsigned int* result,
    double* dev_dataset,
    const double epsilon,
    const unsigned int N,
    const unsigned int DIM
);

// handling data
void importDataset(
    char* fname,
    unsigned int N,
    unsigned int DIM,
    double* dataset
);



int main(int argc, char* argv[])
{
    printf("\nMODE: %d", MODE);
    warmUpGPU();

    char inputFname[500];
    unsigned int N = 0;
    unsigned int DIM = 0;
    double epsilon = 0;


    if (argc != 5)
    {
        fprintf(stderr, "Please provide the following on the command line: N (number of lines in the file), dimensionality (number of coordinates per point), epsilon, dataset filename.\n");
        exit(0);
    }

    sscanf(argv[1], "%d", &N);
    sscanf(argv[2], "%d", &DIM);
    sscanf(argv[3], "%f", &epsilon);
    strcpy(inputFname, argv[4]);

    checkParams(N, DIM);

    printf(
        "\nAllocating the following amount of memory for the dataset: %f GiB",
        (sizeof(double) * N * DIM) / (1024 * 1024 * 1024.0)
    );
    printf(
        "\nAllocating the following amount of memory for the distance matrix: %f GiB",
        (sizeof(double) * N * N) / (1024 * 1024 * 1024.0)
    );


    double tstartbuild = 0.0;
    double tendbuild = 0.0;
    double tstartquery = 0.0;
    double tendquery = 0.0;
    double* dataset = (double*)malloc(sizeof(double) * N * DIM);
    double* distanceMatrix = (double*)malloc(sizeof(double) * N * N);
    unsigned int* result = (unsigned int*)malloc(sizeof(unsigned int) * N);
    importDataset(inputFname, N, DIM, dataset);


    //CPU-only mode
    //It only computes the distance matrix but does not query the distance matrix
    if (MODE == 0)
    {
        double tstartcalc = omp_get_wtime();
        calcDistMatCPU(distanceMatrix, dataset, N, DIM);
        double tendcalc = omp_get_wtime();

        double tstartquery = omp_get_wtime();
        queryDistMat(result, distanceMatrix, epsilon, N);
        double tendquery = omp_get_wtime();

        unsigned int totalWithinEpsilon = 0;
        for (unsigned int i = 0; i < N; i += 1)
        {
            totalWithinEpsilon += result[i];
        }

        printf("\nTotal number of points within epsilon: %u\n", totalWithinEpsilon);
        printf("\nTime to calc the tree: %f", tendcalc - tstartcalc);
        printf("\nTime to query the tree: %f", tendquery - tstartquery);
        printf(
            "\n[MODE: %d, N: %d, E: %0.1f] Total time: %f\n",
            MODE, N, epsilon,
            (tendcalc - tstartcalc) + (tendquery - tstartquery)
        );

        return 0;
    }
    else if (MODE == 1)  // build and query kd-tree on CPU
    {
        tstartbuild = omp_get_wtime();
        struct kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        tendbuild = omp_get_wtime();

        // printf("\n\nkd-tree:");
        // print_tree(tree);
        // printf("\n");

        tstartquery = omp_get_wtime();
        queryKdTreeCPU(&tree, result, dataset, epsilon, N, DIM);
        tendquery = omp_get_wtime();

        unsigned int totalWithinEpsilon = 0;
        for (unsigned int i = 0; i < N; i += 1)
        {
            totalWithinEpsilon += result[i];
        }

        printf("\nTotal number of points within epsilon: %u\n", totalWithinEpsilon);
        printf("\nTime to build the tree: %f", tendbuild - tstartbuild);
        printf("\nTime to query the tree: %f", tendquery - tstartquery);
        printf(
            "\n[MODE: %d, N: %d, E: %0.1f] Total time: %f\n",
            MODE, N, epsilon,
            (tendbuild - tstartbuild) + (tendquery - tstartquery)
        );

        return 0;
    }

    double tstart=omp_get_wtime();

    //Allocate memory for the dataset
    double* dev_dataset;
    gpuErrchk(hipMalloc((double**)&dev_dataset, sizeof(double) * N * DIM));
    gpuErrchk(hipMemcpy(dev_dataset, dataset, sizeof(double) * N * DIM, hipMemcpyHostToDevice));

    //For part 1 that computes the distance matrix
    double* dev_distanceMatrix;
    gpuErrchk(hipMalloc((double**)&dev_distanceMatrix, sizeof(double) * N * N));

    //For part 2 for querying the distance matrix
    unsigned int* resultSet = (unsigned int*)calloc(N, sizeof(unsigned int));
    unsigned int* dev_resultSet;
    gpuErrchk(hipMalloc((unsigned int**)&dev_resultSet, sizeof(unsigned int) * N));
    gpuErrchk(hipMemcpy(dev_resultSet, resultSet, sizeof(unsigned int) * N, hipMemcpyHostToDevice));

    if (MODE == 2)  // brute force with GPU
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N*1.0 / BLOCKDIM*1.0);

        tstartbuild = omp_get_wtime();
        calcDistMatGPU<<<NBLOCKS, BLOCKDIM>>>(dev_distanceMatrix, dev_dataset, N, DIM);
        tendbuild = omp_get_wtime();

        tstartquery = omp_get_wtime();
        queryDistMatGPU<<<NBLOCKS, BLOCKDIM>>>(dev_resultSet, dev_distanceMatrix, epsilon, N);
        tendquery = omp_get_wtime();
    }
    else if (MODE == 3)  // build tree on CPU and query on GPU
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N*1.0 / BLOCKDIM*1.0);
        struct kd_tree_gpu* gpu_tree = NULL;

        tstartbuild = omp_get_wtime();
        struct kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        tendbuild = omp_get_wtime();

        // move the tree onto the GPU - use another pair of time vars to measure the time it takes
        // to move the tree over to the GPU
        //initialize dev gpu node array
	      struct kd_tree_node_gpu* dev_gpu_nodes_array;
       
	      //initialize gpu node array on host
	      //for now, this is just using the an arbritrary size that only works for data set of 100 points;
	      //should figure out a way to calculate gpu node array size based of height of cpu tree;
	      gpu_nodes_array = (struct kd_tree_node_gpu*)malloc(sizeof(struct kd_tree_node_gpu) * 16375);

	      //this is redundant for now; instead of itereting through N, it should probably be through calculated size of gpu node array based on height of the cpu tree
        for(int i = 0; i < N; i++)
	      {
	          init_kd_tree_node_gpu(&(gpu_nodes_array[i]), DIM);
	      }
        
	      //gpuErrchk(hipMalloc((struct kd_tree_node_gpu**)&gpu_nodes_array, sizeof(struct kd_tree_node_gpu) * N));
	      //convert kd tree into heap like structure
	      int max_size = 0;
	      int index_array[N];
	      int index_array_insert = 0;
	      convert_tree_to_array(&(tree->root), &gpu_nodes_array, 0, &max_size, index_array, &index_array_insert);
      	max_size += 1;
        
	      float* dev_data_gpu;
        gpuErrchk(hipMalloc((float**)&(dev_data_gpu), sizeof(float) * DIM));
        
	      //copy over gpu node array from host to device
        //printf("\nleft child index at 16374: %d\n", gpu_nodes_array[16374].left_child_index);
	      gpuErrchk(hipMalloc((struct kd_tree_node_gpu**)&dev_gpu_nodes_array, sizeof(struct kd_tree_node_gpu) * max_size));
        gpuErrchk(hipMemcpy(dev_gpu_nodes_array, gpu_nodes_array, sizeof(struct kd_tree_node_gpu) * max_size, hipMemcpyHostToDevice));
        int insert_index = 0;
	      for(int i = 0; i < N; i++)
	      {
	          insert_index = index_array[i];

	          //copy data from current node in gpu node array to a device data array
	          gpuErrchk(hipMemcpy(dev_data_gpu, gpu_nodes_array[insert_index].data, sizeof(float) * DIM, hipMemcpyHostToDevice));

	          //initialize memory and data on dev gpu node array at the current index; has to be on device code :/
	          init_node_data<<<1, 1>>>(dev_gpu_nodes_array, dev_data_gpu, insert_index);
	      }
	
	      //FOR TESTING AND DEBUGGING FOR NOW
        /*
	      float* testData = (float*)calloc(DIM, sizeof(float));
	      testData[0] = 1337.0;
        printf("\nfirst val in data before: %f\n", testData[0]);
	      printf("last data val in gpu heap: %f\n", gpu_nodes_array[16374].data[0]);
	      float* dev_testData;
        gpuErrchk(hipMalloc((float**)&dev_testData, sizeof(float) * DIM));
        gpuErrchk(hipMemcpy(dev_testData, testData, sizeof(float) * DIM, hipMemcpyHostToDevice));
        queryKdTreeGPU<<<NBLOCKS, BLOCKDIM>>>(dev_gpu_nodes_array, dev_testData);
	      gpuErrchk(hipMemcpy(testData, dev_testData, sizeof(float) * DIM, hipMemcpyDeviceToHost));
	      printf("first val in data after: %f\n", testData[0]);
	      //return 0;
        */
        tstartquery = omp_get_wtime();
        queryKdTreeGPU<<<NBLOCKS, BLOCKDIM>>>(&gpu_tree, dev_resultSet, dev_dataset, epsilon, N, DIM);
        tendquery = omp_get_wtime();
    }
  
    else if (MODE == 4)  // use shared memory
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N*1.0 / BLOCKDIM*1.0);
        struct kd_tree_gpu* gpu_tree = NULL;

        tstartbuild = omp_get_wtime();
        struct kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        tendbuild = omp_get_wtime();

        // move the tree onto the GPU

        tstartquery = omp_get_wtime();
        queryKdTreeGPUWithSharedMem<<<NBLOCKS, BLOCKDIM>>>(&gpu_tree, dev_resultSet, dev_dataset, epsilon, N, DIM);
        tendquery = omp_get_wtime();
    }
    else if (MODE == 5)  // uses 2D block for querying
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N*1.0 / BLOCKDIM*1.0);
        struct kd_tree_gpu* gpu_tree = NULL;


        // build tree on CPU
        tstartbuild = omp_get_wtime();
        kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        tendbuild = omp_get_wtime();

        printf("\nTime to build the tree: %f", tendbuild - tstartbuild);

        printf("\nMODE 5 IS NOT IMPLEMENTED YET!");
        /*
        tstartbuild = omp_get_wtime();
        struct kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        tendbuild = omp_get_wtime();

        // move the tree onto the GPU

        tstartquery = omp_get_wtime();
        queryKdTreeGPUWithTwoDimBlock<<<NBLOCKS, BLOCKDIM>>>(&gpu_tree, dev_resultSet, dev_dataset, epsilon, N, DIM);
        tendquery = omp_get_wtime();
        */
    }
    else if (MODE == 6)  // uses 3D block for querying
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N*1.0 / BLOCKDIM*1.0);
        struct kd_tree_gpu* gpu_tree = NULL;

        printf("\nMODE 6 IS NOT IMPLEMENTED YET!");
        /*
        tstartbuild = omp_get_wtime();
        struct kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        tendbuild = omp_get_wtime();

        // move the tree onto the GPU

        tstartquery = omp_get_wtime();
        queryKdTreeGPUWithThreeDimBlock<<<NBLOCKS, BLOCKDIM>>>(&gpu_tree, dev_resultSet, dev_dataset, epsilon, N, DIM);
        tendquery = omp_get_wtime();
        */
    }

    //Copy result set from the GPU
    gpuErrchk(hipMemcpy(resultSet, dev_resultSet, sizeof(unsigned int) * N, hipMemcpyDeviceToHost));

    //Compute the sum of the result set array
    unsigned int totalWithinEpsilon = 0;

    //Write code here
    for(int resultIndex = 0; resultIndex < N; resultIndex += 1)
    {
        totalWithinEpsilon += resultSet[resultIndex];
    }

    printf("\nTotal number of points within epsilon: %u", totalWithinEpsilon);

    double tend = omp_get_wtime();

    printf("\nTime to build the tree: %f\n", tendbuild - tstartbuild);
    printf("\nTime to query the tree: %f\n", tendquery - tstartquery);

    printf(
        "\n[MODE: %d, N: %d, E: %0.1f] Total time: %f\n",
        MODE, N, epsilon,
        tend - tstart
    );

    printf("\n\n");
    return 0;
}



void warmUpGPU()
{
    printf("\nWarming up GPU for time trialing...\n");
    hipDeviceSynchronize();
    return;
}



void checkParams(unsigned int N, unsigned int DIM)
{
    if (N <= 0 || DIM <= 0)
    {
        fprintf(stderr, "\n Invalid parameters: Error, N: %u, DIM: %u", N, DIM);
        fprintf(stderr, "\nReturning");
        exit(0);
    }
}



void importDataset(
        char* fname,
        unsigned int N,
        unsigned int DIM,
        double* dataset
) {
    FILE *fp = fopen(fname, "r");

    if (!fp)
    {
        fprintf(stderr, "Unable to open file\n");
        fprintf(stderr, "Error: dataset was not imported. Returning.");
        exit(0);
    }

    unsigned int bufferSize = DIM * 10;

    char buf[bufferSize];
    unsigned int rowCnt = 0;
    unsigned int colCnt = 0;
    while (fgets(buf, bufferSize, fp) && rowCnt < N)
    {
        colCnt = 0;

        char *field = strtok(buf, ",");
        double tmp;
        sscanf(field, "%lf", &tmp);

        dataset[rowCnt * DIM + colCnt] = tmp;


        while (field)
        {
            colCnt += 1;
            field = strtok(NULL, ",");

            if (field!=NULL)
            {
                double tmp;
                sscanf(field,"%lf",&tmp);
                dataset[rowCnt*DIM+colCnt]=tmp;
            }
        }

        rowCnt += 1;
    }

    fclose(fp);
}



// cpu
// brute force
void calcDistMatCPU(double* distanceMatrix, const double* dataset, const unsigned int N, const unsigned int DIM)
{
    for (unsigned int i = 0; i < N; i += 1)
    {
        for (unsigned int j = 0; j < N; j += 1)
        {
            double dist = 0.0;

            for (unsigned int d = 0; d < DIM; d += 1)
            {
                dist += (dataset[i * DIM + d] - dataset[j * DIM + d])
                    * (dataset[i * DIM + d] - dataset[j * DIM + d]);
            }

            distanceMatrix[i * N + j] = sqrt(dist);
        }
    }
}


void queryDistMat(unsigned int* result, const double* distanceMatrix, const double epsilon, const unsigned int N)
{
    for (unsigned int i = 0; i < N; i += 1)
    {
        for (unsigned int j = 0; j < N; j += 1)
        {
            if (distanceMatrix[i * N + j] <= epsilon)
            {
                result[i] += 1;
            }
        }
    }
}


// kd-tree
kd_tree_cpu* buildKdTreeCPU(const double* dataset, const unsigned int N, const unsigned int DIM)
{
    kd_tree_cpu* tree;

    init_kd_tree_cpu(&tree);

    for (unsigned int p = 0; p < N; p += 1)
    {
        kd_tree_node_cpu* node;
        double data[DIM];

        for (unsigned int d = 0; d < DIM; d += 1)
        {
            data[d] = dataset[p * DIM + d];
        }

        init_kd_tree_node_cpu(&node, data, DIM, 0);
        insert(&tree, &node);
    }

    return tree;
}


void queryKdTreeCPU(kd_tree_cpu** tree, unsigned int* result, const double* dataset, const double epsilon, const unsigned int N, const unsigned int DIM)
{
    double query[2];
    unsigned int count;
    for (unsigned int p = 0; p < N; p += 1)
    {
        count = 0;

        for (unsigned int d = 0; d < DIM; d += 1)
        {
            query[d] = dataset[p * DIM + d];
        }

        points_within_epsilon(tree, query, epsilon, &count);

        result[p] = count;
    }
}


//===================================================//
//                       GPU                         //
//===================================================//

//querying tree/heap
__global__ void queryKdTreeGPU(struct kd_tree_node_gpu* gpu_nodes_array, float* testData)
{
    //testing/debugging for now
    //gpu_nodes_array[0].level = 69;
    //gpu_nodes_array[0].data[0] = 420;
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;

    if(tid == 0)
    {
	//gpu_nodes_array[0].data = (float*)malloc(sizeof(float) * gpu_nodes_array[0].dim);
        //gpu_nodes_array[0].data[0] = 42069;
	testData[0] = gpu_nodes_array[16374].data[0];
    }

    return;
}

//initialize data pointers on device
__global__ void init_node_data(struct kd_tree_node_gpu* gpu_nodes_array, float* data, int insert_index)
{
    gpu_nodes_array[insert_index].data = (float*)malloc(sizeof(float) * gpu_nodes_array[insert_index].dim);
    for(int i = 0; i < gpu_nodes_array[insert_index].dim; i++)
    {
        gpu_nodes_array[insert_index].data[i] = data[i];
    }
}

// gpu
// brute force?
__global__ void calcDistMatGPU(
    double* distanceMatrix,
    double* dataset,
    const unsigned int N,
    const unsigned int DIM
) {
    const unsigned int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    double dist;


    if (tid >= N)
    {
        return;
    }

    for (unsigned int p = 0; p < N; p += 1)
    {
        dist = 0.0;

        for (unsigned int d = 0; d < DIM; d += 1)
        {
            dist += (dataset[tid * DIM + d] - dataset[p * DIM + d])
                * (dataset[tid * DIM + d] - dataset[p * DIM + d]);
        }

        distanceMatrix[tid * N + i] = sqrt(dist);
    }


    return;
}


__global__ void queryDistMatGPU(
    unsigned int* result,
    double* distanceMatrix,
    const double epsilon,
    const unsigned int N
) {
    const unsigned int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    unsigned int neighbors = 0;

    if (tid >= N)
    {
        return;
    }

    for (unsigned int p = 0; p < N; p += 1)
    {
        if (distanceMatrix[tid * N + p] <= epsilon)
        {
            neighbors += 1;
        }
    }

    result[tid] = neighbors += 1;


    return;
}


// query kd-tree
__global__ void queryKdTreeGPU(
        struct kd_tree_gpu** tree,
        unsigned int* result,
        double* dataset,
        const double epsilon,
        const unsigned int N,
        const unsigned int DIM
) {
    const unsigned int tid = threadIdx.x + (blockIdx.x * blockDim.x);
    double dist = 0.0;
    double dist_prime = 0.0;
    struct kd_tree_node_gpu* working = tree->root;
    struct kd_tree_node_gpu* first = NULL;
    struct kd_tree_node_gpu* second = NULL;

    
    if (tid > N)
    {
        return;
    }
    
    if (query[(*working)->level % (*working)->dim] < (*working)->metric)
    {
        working = &(*working)->left;
    }
    else
    {
        working = &(*working)->right;
    }

    while (1)
    {
        determine_first:
        // 1. determine first and second
        if (query[(*working)->level % (*working)->dim] < (*working)->metric)
        {
            first = &(*working)->left;
            second = &(*working)->right;
        }
        else
        {
            first = &(*working)->right;
            second = &(*working)->left;
        }
        
        // 2. if there is(are) child node(s), determine first, if first is not 'visited'
        if (first != NULL && !first->visited)
        {
            // a2. go to first
            working = first;
        }

        dist_prime = fabsf(query[(*working)->level % (*working)->dim] - (*working)->metric);
        
        // 3. if there is(are) child node(s), determine second, if second is not `visited` - otherwise?
        if (second != NULL && !second->visited && dist_prime < epsilon)
        {
            // a3. go to second
            working = second;
            // b3. go to step 1
            goto determine_first;
        }

        // 4. calc dist
        for (unsigned int i = 0; i < (*node)->dim; i += 1)
        {
            dist += (query[i] - (*node)->data[i])
                        * (query[i] - (*node)->data[i]);
        }
        dist = sqrt(dist);
        // determine if point is within epsilon
        if (dist <= epsilon)
        {
            result[tid] += 1;
        }
        // 5. mark as `visited`
        working->visited = 1;

        // 6. if there is a parent
        if (working->parent != NULL)
        {
            // a6. go to parent
            working = working->parent;
            // b6. go to step 1
        }
        // 7. otherwise, assume tree has been queried
        else
        {
            // a7. break
            break;
        }
    }

    
    return;
}


__global__ void queryKdTreeGPUWithSharedMem(
    struct kd_tree_gpu** tree,
    unsigned int* result,
    double* dev_dataset,
    const double epsilon,
    const unsigned int N,
    const unsigned int DIM
) {
    return;
}
