#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <omp.h>
#include <iostream>
#include <complex.h>
#include <math.h>

#include "../include/kd_tree.cuh"


//Error checking GPU calls
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Mode 0 is CPU brute force - for checking tree results
// Mode 1 is CPU sequential implementation of kd-tree
// Mode 2 brute for query with GPU - for checking and comparing
// Mode 3 uses CPU to build the kd-tree and move to GPU, then uses GPU to query
// Mode 4 uses shared memory to move nodes in question to the tree
// Mode 5 uses 2D block
// Mode 6 uses 3D block
// ...
// #define MODE 0

//Define any constants here
#define BLOCKSIZE 128
//#define NSIZE 10000
using namespace std;


//function prototypes
void warmUpGPU();
void checkParams(unsigned int N, unsigned int DIM);

// cpu code
// brute force
void calcDistMatCPU(
    double* distanceMatrix,
    double* dataset,
    const unsigned int N,
    const unsigned int DIM
);
void queryDistMat(
    unsigned int* result,
    double* distanceMatrix,
    const double epsilon,
    const unsigned int N
);

// kd-tree
kd_tree_cpu* buildKdTreeCPU(const double* dataset, const unsigned int N, const unsigned int DIM);
void queryKdTreeCPU(
    struct kd_tree_cpu** tree,
    unsigned int* result,
    const double* dataset,
    const double epsilon,
    const unsigned int N,
    const unsigned int DIM
);

// gpu code
__global__ void init_node_data(struct kd_tree_node_gpu* gpu_nodes_array, double* data, int insert_index);

// brute force?
__global__ void calcDistMatGPU(
    double* distanceMatrix,
    double* dataset,
    const unsigned int N,
    const unsigned int DIM
);


__global__ void queryDistMatGPU(
    unsigned int* result,
    double* distanceMatrix,
    const double epsilon,
    const unsigned int N
);


// query kd-tree
__global__ void queryKdTreeGPU(
    unsigned int* result,
    struct kd_tree_node_gpu* node_array,
    const double epsilon,
    const unsigned int N,
    const unsigned int DIM
);


__global__ void queryKdTreeGPUWithSharedMem(
    unsigned int* result,
    struct kd_tree_node_gpu* node_array,
    unsigned int* indices,
    const double epsilon,
    const unsigned int N,
    const unsigned int DIM
);



// device
__device__ void zero_seconds(unsigned int* seconds, const unsigned int size);



// handling data
void importDataset(
    char* fname,
    unsigned int N,
    unsigned int DIM,
    double* dataset
);



int main(int argc, char* argv[])
{
    printf("\nMODE: %d", MODE);
    warmUpGPU();

    char inputFname[500];
    unsigned int N = NELEM;
    unsigned int DIM = 0;
    double epsilon = 0;


    if (argc != 4)
    {
        fprintf(stderr, "Please provide the following on the command line: N (number of lines in the file), dimensionality (number of coordinates per point), epsilon, dataset filename.\n");
        exit(0);
    }

    //sscanf(argv[1], "%d", &N);
    sscanf(argv[1], "%d", &DIM);
    sscanf(argv[2], "%f", &epsilon);
    strcpy(inputFname, argv[3]);

    checkParams(N, DIM);

    printf(
        "\nAllocating the following amount of memory for the dataset: %f GiB",
        (sizeof(double) * N * DIM) / (1024 * 1024 * 1024.0)
    );
    printf(
        "\nAllocating the following amount of memory for the distance matrix: %f GiB",
        (sizeof(double) * N * N) / (1024 * 1024 * 1024.0)
    );


    double tstartbuild = 0.0;
    double tendbuild = 0.0;
    double tstartquery = 0.0;
    double tendquery = 0.0;
    double* dataset = (double*)malloc(sizeof(double) * N * DIM);
    double* distanceMatrix = (double*)malloc(sizeof(double) * N * N);
    unsigned int* result = (unsigned int*)malloc(sizeof(unsigned int) * N);
    importDataset(inputFname, N, DIM, dataset);


    //CPU-only mode
    //It only computes the distance matrix but does not query the distance matrix
    if (MODE == 0)
    {
        double tstartcalc = omp_get_wtime();
        calcDistMatCPU(distanceMatrix, dataset, N, DIM);
        double tendcalc = omp_get_wtime();

        double tstartquery = omp_get_wtime();
        queryDistMat(result, distanceMatrix, epsilon, N);
        double tendquery = omp_get_wtime();

        unsigned int totalWithinEpsilon = 0;
        for (unsigned int i = 0; i < N; i += 1)
        {
            totalWithinEpsilon += result[i];
        }

        printf("\nTotal number of points within epsilon: %u\n", totalWithinEpsilon);
        printf("\nTime to calc the tree: %f", tendcalc - tstartcalc);
        printf("\nTime to query the tree: %f", tendquery - tstartquery);
        printf(
            "\n[MODE: %d, N: %d, E: %0.1f] Total time: %f\n",
            MODE, N, epsilon,
            (tendcalc - tstartcalc) + (tendquery - tstartquery)
        );

        return 0;
    }
    else if (MODE == 1)  // build and query kd-tree on CPU
    {
        tstartbuild = omp_get_wtime();
        struct kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        tendbuild = omp_get_wtime();

        // printf("\n\nkd-tree:");
        // print_tree(tree);
        // printf("\n");

        tstartquery = omp_get_wtime();
        queryKdTreeCPU(&tree, result, dataset, epsilon, N, DIM);
        tendquery = omp_get_wtime();

        unsigned int totalWithinEpsilon = 0;
        for (unsigned int i = 0; i < N; i += 1)
        {
            totalWithinEpsilon += result[i];
        }

        printf("\nTotal number of points within epsilon: %u\n", totalWithinEpsilon);
        printf("\nTime to build the tree: %f", tendbuild - tstartbuild);
        printf("\nTime to query the tree: %f", tendquery - tstartquery);
        printf(
            "\n[MODE: %d, N: %d, E: %0.1f] Total time: %f\n",
            MODE, N, epsilon,
            (tendbuild - tstartbuild) + (tendquery - tstartquery)
        );

        return 0;
    }

    double tstart = omp_get_wtime();

    //Allocate memory for the dataset
    double* dev_dataset;
    gpuErrchk(hipMalloc((double**)&dev_dataset, sizeof(double) * N * DIM));
    gpuErrchk(hipMemcpy(dev_dataset, dataset, sizeof(double) * N * DIM, hipMemcpyHostToDevice));

    //For part 1 that computes the distance matrix
    double* dev_distanceMatrix;
    gpuErrchk(hipMalloc((double**)&dev_distanceMatrix, sizeof(double) * N * N));

    //For part 2 for querying the distance matrix
    unsigned int* resultSet = (unsigned int*)calloc(N, sizeof(unsigned int));
    unsigned int* dev_resultSet;
    gpuErrchk(hipMalloc((unsigned int**)&dev_resultSet, sizeof(unsigned int) * N));
    gpuErrchk(hipMemcpy(dev_resultSet, resultSet, sizeof(unsigned int) * N, hipMemcpyHostToDevice));

    if (MODE == 2)  // brute force with GPU
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N*1.0 / BLOCKDIM*1.0);

        tstartbuild = omp_get_wtime();
        calcDistMatGPU<<<NBLOCKS, BLOCKDIM>>>(dev_distanceMatrix, dev_dataset, N, DIM);
        tendbuild = omp_get_wtime();

        tstartquery = omp_get_wtime();
        queryDistMatGPU<<<NBLOCKS, BLOCKDIM>>>(dev_resultSet, dev_distanceMatrix, epsilon, N);
        tendquery = omp_get_wtime();
    }
    else if (MODE == 3)  // build tree on CPU and query on GPU
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N*1.0 / BLOCKDIM*1.0);
        struct kd_tree_gpu* gpu_tree = NULL;

        tstartbuild = omp_get_wtime();
        struct kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        tendbuild = omp_get_wtime();

        // move the tree onto the GPU - use another pair of time vars to measure the time it takes
        // to move the tree over to the GPU
        // initialize dev gpu node array
        struct kd_tree_node_gpu* dev_gpu_nodes_array;
       
        // initialize gpu node array on host
        // calculate gpu node array size based of height of cpu tree;
	set_tree_height(tree->root, &(tree->height));
	unsigned int gpu_node_array_size = get_array_size(tree->height);
        //struct kd_tree_node_gpu* gpu_nodes_array = (struct kd_tree_node_gpu*)malloc(sizeof(struct kd_tree_node_gpu) * gpu_node_array_size);
        struct kd_tree_node_gpu* gpu_nodes_array = (struct kd_tree_node_gpu*)malloc(sizeof(struct kd_tree_node_gpu) * N);
        //initialize gpu array
        for(unsigned int i = 0; i < N; i++)
        {
            init_kd_tree_node_gpu(&(gpu_nodes_array[i]), DIM);
        }
        
        // gpuErrchk(hipMalloc((struct kd_tree_node_gpu**)&gpu_nodes_array, sizeof(struct kd_tree_node_gpu) * N));
        // convert kd tree into heap like structure
	unsigned int index_insert = 0;
        convert_tree_to_array(&(tree->root), &gpu_nodes_array, &index_insert);
        //for(int i = 0; i < N; i++)
	//{
        //    printf("\ncurrent node metric: %lf", gpu_nodes_array[i].metric);
	//}

        double* dev_data_gpu;
        gpuErrchk(hipMalloc((double**)&(dev_data_gpu), sizeof(double) * DIM));
        
        // copy over gpu node array from host to device
        gpuErrchk(hipMalloc((struct kd_tree_node_gpu**)&dev_gpu_nodes_array, sizeof(struct kd_tree_node_gpu) * N));
        gpuErrchk(hipMemcpy(dev_gpu_nodes_array, gpu_nodes_array, sizeof(struct kd_tree_node_gpu) * N, hipMemcpyHostToDevice));
        for(unsigned int i = 0; i < N; i++)
        {
            // copy data from current node in gpu node array to a device data array
            gpuErrchk(hipMemcpy(dev_data_gpu, gpu_nodes_array[i].data, sizeof(double) * DIM, hipMemcpyHostToDevice));

            // initialize memory and data on dev gpu node array at the current index; has to be on device code :/
            init_node_data<<<1, 1>>>(dev_gpu_nodes_array, dev_data_gpu, i);
        }

        //begin querying
        tstartquery = omp_get_wtime();
        queryKdTreeGPU<<<NBLOCKS, BLOCKDIM>>>(dev_resultSet, dev_gpu_nodes_array, epsilon, N, DIM);
        tendquery = omp_get_wtime();
    }
    else if (MODE == 4)  // use shared memory
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N*1.0 / BLOCKDIM*1.0);
        struct kd_tree_gpu* gpu_tree = NULL;

        printf("\nMODE 4 IS NOT IMPLEMENTED YET!");
        /*
        tstartbuild = omp_get_wtime();
        struct kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        tendbuild = omp_get_wtime();

        // move the tree onto the GPU

        tstartquery = omp_get_wtime();
        queryKdTreeGPUWithSharedMem<<<NBLOCKS, BLOCKDIM>>>(&gpu_tree, dev_resultSet, dev_dataset, epsilon, N, DIM);
        tendquery = omp_get_wtime();
        */
    }
    else if (MODE == 5)  // uses 2D block for querying
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N*1.0 / BLOCKDIM*1.0);
        struct kd_tree_gpu* gpu_tree = NULL;


        // build tree on CPU
        tstartbuild = omp_get_wtime();
        kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        tendbuild = omp_get_wtime();

        printf("\nTime to build the tree: %f", tendbuild - tstartbuild);

        printf("\nMODE 5 IS NOT IMPLEMENTED YET!");
        /*
        tstartbuild = omp_get_wtime();
        struct kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        tendbuild = omp_get_wtime();

        // move the tree onto the GPU

        tstartquery = omp_get_wtime();
        queryKdTreeGPUWithTwoDimBlock<<<NBLOCKS, BLOCKDIM>>>(&gpu_tree, dev_resultSet, dev_dataset, epsilon, N, DIM);
        tendquery = omp_get_wtime();
        */
    }
    else if (MODE == 6)  // uses 3D block for querying
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N*1.0 / BLOCKDIM*1.0);
        struct kd_tree_gpu* gpu_tree = NULL;

        printf("\nMODE 6 IS NOT IMPLEMENTED YET!");
        /*
        tstartbuild = omp_get_wtime();
        struct kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        tendbuild = omp_get_wtime();

        // move the tree onto the GPU

        tstartquery = omp_get_wtime();
        queryKdTreeGPUWithThreeDimBlock<<<NBLOCKS, BLOCKDIM>>>(&gpu_tree, dev_resultSet, dev_dataset, epsilon, N, DIM);
        tendquery = omp_get_wtime();
        */
    }

    //Copy result set from the GPU
    gpuErrchk(hipMemcpy(resultSet, dev_resultSet, sizeof(unsigned int) * N, hipMemcpyDeviceToHost));

    //Compute the sum of the result set array
    unsigned int totalWithinEpsilon = 0;

    //Write code here
    for(int resultIndex = 0; resultIndex < N; resultIndex += 1)
    {
        totalWithinEpsilon += resultSet[resultIndex];
    }

    printf("\nTotal number of points within epsilon: %u", totalWithinEpsilon);

    double tend = omp_get_wtime();

    printf("\nTime to build the tree: %f\n", tendbuild - tstartbuild);
    printf("\nTime to query the tree: %f\n", tendquery - tstartquery);

    printf(
        "\n[MODE: %d, N: %d, E: %0.1f] Total time: %f\n",
        MODE, N, epsilon,
        tend - tstart
    );

    printf("\n\n");
    return 0;
}



void warmUpGPU()
{
    printf("\nWarming up GPU for time trialing...\n");
    hipDeviceSynchronize();
    return;
}


void checkParams(unsigned int N, unsigned int DIM)
{
    if (DIM <= 0)
    {
        fprintf(stderr, "\n Invalid parameters: Error, DIM: %u", DIM);
        fprintf(stderr, "\nReturning");
        exit(0);
    }
}


void importDataset(
        char* fname,
        unsigned int N,
        unsigned int DIM,
        double* dataset
) {
    FILE *fp = fopen(fname, "r");

    if (!fp)
    {
        fprintf(stderr, "Unable to open file\n");
        fprintf(stderr, "Error: dataset was not imported. Returning.");
        exit(0);
    }

    unsigned int bufferSize = DIM * 10;

    char buf[bufferSize];
    unsigned int rowCnt = 0;
    unsigned int colCnt = 0;
    while (fgets(buf, bufferSize, fp) && rowCnt < N)
    {
        colCnt = 0;

        char *field = strtok(buf, ",");
        double tmp;
        sscanf(field, "%lf", &tmp);

        dataset[rowCnt * DIM + colCnt] = tmp;


        while (field)
        {
            colCnt += 1;
            field = strtok(NULL, ",");

            if (field!=NULL)
            {
                double tmp;
                sscanf(field,"%lf",&tmp);
                dataset[rowCnt*DIM+colCnt]=tmp;
            }
        }

        rowCnt += 1;
    }

    fclose(fp);
}



// cpu
// brute force
void calcDistMatCPU(double* distanceMatrix, const double* dataset, const unsigned int N, const unsigned int DIM)
{
    for (unsigned int i = 0; i < N; i += 1)
    {
        for (unsigned int j = 0; j < N; j += 1)
        {
            double dist = 0.0;

            for (unsigned int d = 0; d < DIM; d += 1)
            {
                dist += (dataset[i * DIM + d] - dataset[j * DIM + d])
                    * (dataset[i * DIM + d] - dataset[j * DIM + d]);
            }

            distanceMatrix[i * N + j] = sqrt(dist);
        }
    }
}


void queryDistMat(unsigned int* result, const double* distanceMatrix, const double epsilon, const unsigned int N)
{
    for (unsigned int i = 0; i < N; i += 1)
    {
        for (unsigned int j = 0; j < N; j += 1)
        {
            if (distanceMatrix[i * N + j] <= epsilon)
            {
                result[i] += 1;
            }
        }
    }
}


// kd-tree
kd_tree_cpu* buildKdTreeCPU(const double* dataset, const unsigned int N, const unsigned int DIM)
{
    kd_tree_cpu* tree;

    init_kd_tree_cpu(&tree);

    for (unsigned int p = 0; p < N; p += 1)
    {
        kd_tree_node_cpu* node;
        double data[DIM];

        for (unsigned int d = 0; d < DIM; d += 1)
        {
            data[d] = dataset[p * DIM + d];
        }

        init_kd_tree_node_cpu(&node, data, DIM);
        insert(&tree, &node);
    }

    return tree;
}


void queryKdTreeCPU(kd_tree_cpu** tree, unsigned int* result, const double* dataset, const double epsilon, const unsigned int N, const unsigned int DIM)
{
    double query[2];
    unsigned int count;
    for (unsigned int p = 0; p < N; p += 1)
    {
        count = 0;

        for (unsigned int d = 0; d < DIM; d += 1)
        {
            query[d] = dataset[p * DIM + d];
        }

        points_within_epsilon_cpu(tree, query, epsilon, &count);

        result[p] = count;
    }
}



//===================================================//
//                       GPU                         //
//===================================================//

//querying tree/heap
__global__ void queryKdTreeGPU(struct kd_tree_node_gpu* gpu_nodes_array, double* testData)
{
    //testing/debugging for now
    //gpu_nodes_array[0].level = 69;
    //gpu_nodes_array[0].data[0] = 420;
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;

    if(tid == 0)
    {
	//gpu_nodes_array[0].data = (double*)malloc(sizeof(double) * gpu_nodes_array[0].dim);
        //gpu_nodes_array[0].data[0] = 42069;
	testData[0] = gpu_nodes_array[16374].data[0];
    }

    return;
}


//initialize data pointers on device
__global__ void init_node_data(struct kd_tree_node_gpu* gpu_nodes_array, double* data, int insert_index)
{
    gpu_nodes_array[insert_index].data = (double*)malloc(sizeof(double) * gpu_nodes_array[insert_index].dim);
    for(int i = 0; i < gpu_nodes_array[insert_index].dim; i++)
    {
        gpu_nodes_array[insert_index].data[i] = data[i];
    }
}


// brute force?
__global__ void calcDistMatGPU(
    double* distanceMatrix,
    double* dataset,
    const unsigned int N,
    const unsigned int DIM
) {
    const unsigned int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    double dist;


    if (tid >= N)
    {
        return;
    }

    for (unsigned int p = 0; p < N; p += 1)
    {
        dist = 0.0;

        for (unsigned int d = 0; d < DIM; d += 1)
        {
            dist += (dataset[tid * DIM + d] - dataset[p * DIM + d])
                * (dataset[tid * DIM + d] - dataset[p * DIM + d]);
        }

        distanceMatrix[tid * N + p] = sqrt(dist);
    }


    return;
}


__global__ void queryDistMatGPU(
    unsigned int* result,
    double* distanceMatrix,
    const double epsilon,
    const unsigned int N
) {
    const unsigned int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    unsigned int neighbors = 0;

    if (tid >= N)
    {
        return;
    }

    for (unsigned int p = 0; p < N; p += 1)
    {
        if (distanceMatrix[tid * N + p] <= epsilon)
        {
            neighbors += 1;
        }
    }

    result[tid] = neighbors += 1;


    return;
}


// query kd-tree
__global__ void queryKdTreeGPU(
        unsigned int* result,
        struct kd_tree_node_gpu* node_array,
        const double epsilon,
        const unsigned int N,
        const unsigned int DIM
) {
    const unsigned int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    
    if (tid >= N)
    {
        return;
    }

    double dist = 0.0;
    double dist_prime = 0.0;
    unsigned int count = 0;
    int first_index = 0;
    int second_index = 0;
    //printf("\n\ngrabbing query...");
    //double* query = node_array[indices[tid]].data;
    double* query = node_array[tid].data;
    //printf("\ngrabbed query!");
    struct kd_tree_node_gpu* working = &node_array[0];

    // allcate space to store seconds which must be visited
    const unsigned int NUM_SECONDS = N;

    //have to use constant NELEM in order to do this
    //the alternative of dynamic memory breaks with large N
    unsigned int seconds[NELEM];
    //unsigned int* seconds = (unsigned int*)malloc(sizeof(unsigned int) * NUM_SECONDS);  // guess of how many times the second will be visited
    zero_seconds(seconds, NUM_SECONDS);

    // secondary index for entering new points in 'seconds' called 's'
    unsigned int s = 0;
    unsigned int i = 0;
    // loop over seconds array
    while (i < NUM_SECONDS)
    {
        // label: visit_subtree
        visit_subtree:

        // loop until end of tree
        //printf("\n\nvisiting a sub-tree...");
        while (working != NULL)
        {
            dist = 0.0;
            dist_prime = 0.0;
            // 1. calc dist
            //printf("\ncalculating distance...");
            for (unsigned int d = 0; d < DIM; d += 1)
            {
                dist += (query[d] - working->data[d]) * (query[d] - working->data[d]);
            }
            dist = sqrt(dist);
            //printf("\nfinised calculating distance!");

            // 2. check point within 'epsilon'
            if (dist <= epsilon)
            {
                // 2a. update result
                count += 1;
            }

            // 3. check query less than metric
            //printf("\ndetermining first and second nodes...");
            if (query[working->level % DIM] < working->metric)
            {
                // 3a. set first to left
                first_index = working->left_child_index;
                // 3b. set second to right
                second_index = working->right_child_index;
            }
            // 4. otherwise, assume query greater than metric
            else
            {
                // 4a. set first to right
                first_index = working->right_child_index;
                // 4b. set second to left
                second_index = working->left_child_index;
            }
            //printf("\nfinished determining first and seconds nodes!");

            // 5. calc dist to split axis
            dist_prime = fabsf(query[working->level % DIM] - working->metric);

            // 6. check second exists and check split axis within 'epsilon'
            //printf("\n\nsaving and updating index 's'...");
            if (s < NUM_SECONDS && second_index > 0 && dist_prime < epsilon)
            {
                // 6a. save second at 's'
                seconds[s] = second_index;
                // 6b. update 's'
                s += 1;
                // 6c. set second at 's' to 0
            }
            //printf("\nfinished saving second and updating index 's'!");

            // 7. set workin->to first
            //printf("\n\nmoving 'working' to node at 'first_index'...");
	    if(first_index != -1)
	    {
                working = &node_array[first_index];
	    }

	    else
	    {
                working = NULL;
	    }
            //printf("\nfinished moving 'working' to node at 'first_index'!");
        }
        //printf("\nfinished visiting sub-tree!");

        // 8. check need to visit a second
        /*
        printf("\n\nreaching into 'seconds' at 'i'...");
        seconds[i] = 0;
        printf("\nfinished reaching into 'seconds' at 'i' and setting value to 0!");
        */
        if (i < NUM_SECONDS && seconds[i] > 0)
        {
            // 8a. set working to second at 'i' in seconds
            working = &node_array[seconds[i]];
            // 8b. update 'i'
            i += 1;
            // 8c. go to label 'visit_subtree'
            goto visit_subtree;
        }
        // 9. otherwise, assume query is finished
        else
        {
            // 9a. break loop
            break;
        }
    }

    //printf("\n\nupdating result at 'tid'...");
    result[tid] = count;
    //printf("\nfinished updating result at 'tid'!");
 
    
    return;
}


__global__ void queryKdTreeGPUWithSharedMem(
    struct kd_tree_node_gpu** node_array,
    unsigned int* result,
    const double epsilon,
    const unsigned int N,
    const unsigned int DIM
) {
    return;
}



// device
__device__ void zero_seconds(unsigned int* seconds, const unsigned int size)
{
    for (unsigned int i = 0; i < size; i += 1)
    {
        seconds[i] = 0;
    }
}
