#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <omp.h>
#include <iostream>
#include <complex.h>
#include <math.h>

#include "../include/kd_tree.cuh"


//Error checking GPU calls
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Mode 0 is CPU brute force - for checking tree results
// Mode 1 is CPU sequential implementation of kd-tree
// Mode 2 brute for query with GPU - for checking and comparing
// Mode 3 uses CPU to build the kd-tree and GPU to query
// Mode 4 ...
// ...
// #define MODE 0

//Define any constants here
#define BLOCKSIZE 128

using namespace std;


//function prototypes
void warmUpGPU();
void checkParams(unsigned int N, unsigned int DIM);

// cpu code
// brute force
void calcDistMatCPU(double* distanceMatrix, const double* dataset, const unsigned int N, const unsigned int DIM);
void queryDistMat(unsigned int* result, const double* distanceMatrix, const double epsilon, const unsigned int N);

// kd-tree
kd_tree_cpu* buildKdTreeCPU(const double* dataset, const unsigned int N, const unsigned int DIM);
void queryKdTreeCPU(struct kd_tree_cpu** tree, unsigned int* result, const double* dataset, const double epsilon, const unsigned int N, const unsigned int DIM);

// gpu code
// brute force?

// kd-tree
__global__ void queryKdTreeGPU(struct kd_tree_gpu** tree, unsigned int* result, double* dataset, const double epsilon, const unsigned int N, const unsigned int DIM);

// handling data
void importDataset(
    char* fname,
    unsigned int N,
    unsigned int DIM,
    double* dataset
);



int main(int argc, char* argv[])
{
    printf("\nMODE: %d", MODE);
    warmUpGPU();

    char inputFname[500];
    unsigned int N = 0;
    unsigned int DIM = 0;
    double epsilon = 0;


    if (argc != 5)
    {
        fprintf(stderr, "Please provide the following on the command line: N (number of lines in the file), dimensionality (number of coordinates per point), epsilon, dataset filename.\n");
        exit(0);
    }

    sscanf(argv[1], "%d", &N);
    sscanf(argv[2], "%d", &DIM);
    sscanf(argv[3], "%f", &epsilon);
    strcpy(inputFname, argv[4]);

    checkParams(N, DIM);

    printf(
        "\nAllocating the following amount of memory for the dataset: %f GiB",
        (sizeof(double) * N * DIM) / (1024 * 1024 * 1024.0)
    );
    printf(
        "\nAllocating the following amount of memory for the distance matrix: %f GiB",
        (sizeof(double) * N * N) / (1024 * 1024 * 1024.0)
    );


    double tstartbuild = 0.0;
    double tendbuild = 0.0;
    double tstartquery = 0.0;
    double tendquery = 0.0;
    double* dataset = (double*)malloc(sizeof(double) * N * DIM);
    double* distanceMatrix = (double*)malloc(sizeof(double) * N * N);
    unsigned int* result = (unsigned int*)malloc(sizeof(unsigned int) * N);
    importDataset(inputFname, N, DIM, dataset);


    //CPU-only mode
    //It only computes the distance matrix but does not query the distance matrix
    if (MODE == 0)
    {
        double tstartcalc = omp_get_wtime();
        calcDistMatCPU(distanceMatrix, dataset, N, DIM);
        double tendcalc = omp_get_wtime();

        double tstartquery = omp_get_wtime();
        queryDistMat(result, distanceMatrix, epsilon, N);
        double tendquery = omp_get_wtime();

        unsigned int totalWithinEpsilon = 0;
        for (unsigned int i = 0; i < N; i += 1)
        {
            totalWithinEpsilon += result[i];
        }

        printf("\nTotal number of points within epsilon: %u\n", totalWithinEpsilon);
        printf("\nTime to calc the tree: %f", tendcalc - tstartcalc);
        printf("\nTime to query the tree: %f", tendquery - tstartquery);
        printf(
            "\n[MODE: %d, N: %d, E: %0.1f] Total time: %f\n",
            MODE, N, epsilon,
            (tendcalc - tstartcalc) + (tendquery - tstartquery)
        );

        return 0;
    }
    else if (MODE == 1)  // build and query kd-tree on CPU
    {
        tstartbuild = omp_get_wtime();
        struct kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        tendbuild = omp_get_wtime();

        // printf("\n\nkd-tree:");
        // print_tree(tree);
        // printf("\n");

        tstartquery = omp_get_wtime();
        queryKdTreeCPU(&tree, result, dataset, epsilon, N, DIM);
        tendquery = omp_get_wtime();

        unsigned int totalWithinEpsilon = 0;
        for (unsigned int i = 0; i < N; i += 1)
        {
            totalWithinEpsilon += result[i];
        }

        printf("\nTotal number of points within epsilon: %u\n", totalWithinEpsilon);
        printf("\nTime to build the tree: %f", tendbuild - tstartbuild);
        printf("\nTime to query the tree: %f", tendquery - tstartquery);
        printf(
            "\n[MODE: %d, N: %d, E: %0.1f] Total time: %f\n",
            MODE, N, epsilon,
            (tendbuild - tstartbuild) + (tendquery - tstartquery)
        );

        return 0;
    }

    double tstart=omp_get_wtime();

    //Allocate memory for the dataset
    double* dev_dataset;
    gpuErrchk(hipMalloc((double**)&dev_dataset, sizeof(double) * N * DIM));
    gpuErrchk(hipMemcpy(dev_dataset, dataset, sizeof(double) * N * DIM, hipMemcpyHostToDevice));

    //For part 1 that computes the distance matrix
    double* dev_distanceMatrix;
    gpuErrchk(hipMalloc((double**)&dev_distanceMatrix, sizeof(double) * N * N));

    //For part 2 for querying the distance matrix
    unsigned int* resultSet = (unsigned int*)calloc(N, sizeof(unsigned int));
    unsigned int* dev_resultSet;
    gpuErrchk(hipMalloc((unsigned int**)&dev_resultSet, sizeof(unsigned int) * N));
    gpuErrchk(hipMemcpy(dev_resultSet, resultSet, sizeof(unsigned int) * N, hipMemcpyHostToDevice));

    if (MODE == 2)  // brute force with GPU
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N * 1.0 / BLOCKDIM);

        // calculate distance matrix

        // query distance matrix
    }
    else if (MODE == 3)  // build tree on CPU and query on GPU
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N * 1.0 / BLOCKDIM);
        struct kd_tree_gpu* gpu_tree = NULL;

        tstartbuild = omp_get_wtime();
        struct kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        tendbuild = omp_get_wtime();

        // move the tree onto the GPU

        tstartquery = omp_get_wtime();
        queryKdTreeGPU<<<NBLOCKS, BLOCKDIM>>>(&gpu_tree, dev_resultSet, dev_dataset, epsilon, N, DIM);
        tendquery = omp_get_wtime();
    }

    //Copy result set from the GPU
    gpuErrchk(hipMemcpy(resultSet, dev_resultSet, sizeof(unsigned int) * N, hipMemcpyDeviceToHost));

    //Compute the sum of the result set array
    unsigned int totalWithinEpsilon = 0;

    //Write code here
    for(int resultIndex = 0; resultIndex < N; resultIndex += 1)
    {
        totalWithinEpsilon += resultSet[resultIndex];
    }

    printf("\nTotal number of points within epsilon: %u", totalWithinEpsilon);

    double tend = omp_get_wtime();

    printf("\nTime to build the tree: %f\n", tendbuild - tstartbuild);
    printf("\nTime to query the tree: %f\n", tendquery - tstartquery);

    printf(
        "\n[MODE: %d, N: %d, E: %0.1f] Total time: %f\n",
        MODE, N, epsilon,
        tend - tstart
    );

    printf("\n\n");
    return 0;
}



void warmUpGPU()
{
    printf("\nWarming up GPU for time trialing...\n");
    hipDeviceSynchronize();
    return;
}



void checkParams(unsigned int N, unsigned int DIM)
{
    if (N <= 0 || DIM <= 0)
    {
        fprintf(stderr, "\n Invalid parameters: Error, N: %u, DIM: %u", N, DIM);
        fprintf(stderr, "\nReturning");
        exit(0);
    }
}



void importDataset(
        char* fname,
        unsigned int N,
        unsigned int DIM,
        double* dataset
) {
    FILE *fp = fopen(fname, "r");

    if (!fp)
    {
        fprintf(stderr, "Unable to open file\n");
        fprintf(stderr, "Error: dataset was not imported. Returning.");
        exit(0);
    }

    unsigned int bufferSize = DIM * 10;

    char buf[bufferSize];
    unsigned int rowCnt = 0;
    unsigned int colCnt = 0;
    while (fgets(buf, bufferSize, fp) && rowCnt < N)
    {
        colCnt = 0;

        char *field = strtok(buf, ",");
        double tmp;
        sscanf(field, "%lf", &tmp);

        dataset[rowCnt * DIM + colCnt] = tmp;


        while (field)
        {
            colCnt += 1;
            field = strtok(NULL, ",");

            if (field!=NULL)
            {
                double tmp;
                sscanf(field,"%lf",&tmp);
                dataset[rowCnt*DIM+colCnt]=tmp;
            }
        }

        rowCnt += 1;
    }

    fclose(fp);
}



// cpu
// brute force
void calcDistMatCPU(double* distanceMatrix, const double* dataset, const unsigned int N, const unsigned int DIM)
{
    for (unsigned int i = 0; i < N; i += 1)
    {
        for (unsigned int j = 0; j < N; j += 1)
        {
            double dist = 0.0;

            for (unsigned int d = 0; d < DIM; d += 1)
            {
                dist += (dataset[i * DIM + d] - dataset[j * DIM + d])
                    * (dataset[i * DIM + d] - dataset[j * DIM + d]);
            }

            distanceMatrix[i * N + j] = sqrt(dist);
        }
    }
}


void queryDistMat(unsigned int* result, const double* distanceMatrix, const double epsilon, const unsigned int N)
{
    for (unsigned int i = 0; i < N; i += 1)
    {
        for (unsigned int j = 0; j < N; j += 1)
        {
            if (distanceMatrix[i * N + j] <= epsilon)
            {
                result[i] += 1;
            }
        }
    }
}


// kd-tree
kd_tree_cpu* buildKdTreeCPU(const double* dataset, const unsigned int N, const unsigned int DIM)
{
    kd_tree_cpu* tree;

    init_kd_tree_cpu(&tree);

    for (unsigned int p = 0; p < N; p += 1)
    {
        kd_tree_node_cpu* node;
        double data[DIM];

        for (unsigned int d = 0; d < DIM; d += 1)
        {
            data[d] = dataset[p * DIM + d];
        }

        init_kd_tree_node_cpu(&node, data, DIM, 0);
        insert(&tree, &node);
    }

    return tree;
}


void queryKdTreeCPU(kd_tree_cpu** tree, unsigned int* result, const double* dataset, const double epsilon, const unsigned int N, const unsigned int DIM)
{
    double query[2];
    unsigned int count;
    for (unsigned int p = 0; p < N; p += 1)
    {
        count = 0;

        for (unsigned int d = 0; d < DIM; d += 1)
        {
            query[d] = dataset[p * DIM + d];
        }

        points_within_epsilon(tree, query, epsilon, &count);

        result[p] = count;
    }
}



// gpu
// brute force?


// kd-tree
__global__ void queryKdTreeGPU(struct kd_tree_gpu** tree, unsigned int* result, double* dataset, const double epsilon, const unsigned int N, const unsigned int DIM)
{
    const unsigned int tid = threadIdx.x + (blockIdx.x * blockDim.x);
    double dist = 0.0;
    double dist_prime = 0.0;
    struct kd_tree_node_gpu* working = tree->root;
    struct kd_tree_node_gpu* first = NULL;
    struct kd_tree_node_gpu* second = NULL;

    
    if (tid > N)
    {
        return;
    }
    
    // 1. choose first

    // loop - inf loop
    {
        // 1. determine first and second
        
        // 2. if there is(are) child node(s), determine first, if first is not 'visited'
        {
            // a2. go to first
            // b2. go to step 1 - maybe not necessary if coming from bottom
        }
        
        // 3. if there is(are) child node(s), determine second, if second is not `visited`
        {
            // a3. go to second
            // b3. go to step 1
        }

        // 4. calc dist
        // 5. mark as `visited`

        // 6. if there is a parent
        {
            // a6. go to parent
            // b6. go to step 1
        }
        // 7. otherwise, assume tree has been queried
        {
            // a7. break
        }
    }

    
    return 0;
}
