#include "hip/hip_runtime.h"
<<<<<<< HEAD
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <omp.h>
#include <iostream>
#include <complex.h>
#include <math.h>

#include "../include/kd_tree.cuh"


//Error checking GPU calls
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Mode 0 is CPU brute force - for checking tree results
// Mode 1 is CPU sequential implementation of kd-tree
// Mode 2 brute for query with GPU - for checking and comparing
// Mode 3 uses CPU to build the kd-tree and GPU to query
// Mode 4 ...
// ...
// #define MODE 0

//Define any constants here
#define BLOCKSIZE 128

using namespace std;


//function prototypes
void warmUpGPU();
void checkParams(unsigned int N, unsigned int DIM);

// cpu code
// brute force
void calcDistMatCPU(float* distanceMatrix, const float* dataset, const unsigned int N, const unsigned int DIM);
void queryDistMat(unsigned int* result, const float* distanceMatrix, const float epsilon, const unsigned int N);

// kd-tree
kd_tree_cpu* buildKdTreeCPU(const float* dataset, const unsigned int N, const unsigned int DIM);
void queryKdTreeCPU(kd_tree_cpu** tree, unsigned int* result, const float* dataset, const float epsilon, const unsigned int N, const unsigned int DIM);

// gpu code

// handling data
void importDataset(
    char* fname,
    unsigned int N,
    unsigned int DIM,
    float* dataset
);



int main(int argc, char* argv[])
{
    printf("\nMODE: %d", MODE);
    warmUpGPU();

    char inputFname[500];
    unsigned int N = 0;
    unsigned int DIM = 0;
    float epsilon = 0;


    if (argc != 5)
    {
        fprintf(stderr, "Please provide the following on the command line: N (number of lines in the file), dimensionality (number of coordinates per point), epsilon, dataset filename.\n");
        exit(0);
    }

    sscanf(argv[1], "%d", &N);
    sscanf(argv[2], "%d", &DIM);
    sscanf(argv[3], "%f", &epsilon);
    strcpy(inputFname,argv[4]);

    checkParams(N, DIM);

    printf(
        "\nAllocating the following amount of memory for the dataset: %f GiB",
        (sizeof(float) * N * DIM) / (1024 * 1024 * 1024.0)
    );
    printf(
        "\nAllocating the following amount of memory for the distance matrix: %f GiB",
        (sizeof(float) * N * N) / (1024 * 1024 * 1024.0)
    );


    double tstartbuild = 0.0;
    double tendbuild = 0.0;
    double tstartquery = 0.0;
    double tendquery = 0.0;
    float* dataset = (float*)malloc(sizeof(float) * N * DIM);
    float* distanceMatrix = (float*)malloc(sizeof(float) * N * N);
    unsigned int* result = (unsigned int*)malloc(sizeof(unsigned int) * N);
    importDataset(inputFname, N, DIM, dataset);


    //CPU-only mode
    //It only computes the distance matrix but does not query the distance matrix
    if (MODE == 0)
    {
        double tstartcalc = omp_get_wtime();
        calcDistMatCPU(distanceMatrix, dataset, N, DIM);
        double tendcalc = omp_get_wtime();

        double tstartquery = omp_get_wtime();
        queryDistMat(result, distanceMatrix, epsilon, N);
        double tendquery = omp_get_wtime();

        unsigned int totalWithinEpsilon = 0;
        for (unsigned int i = 0; i < N; i += 1)
        {
            totalWithinEpsilon += result[i];
        }

        printf("\nTotal number of points within epsilon: %u\n", totalWithinEpsilon);
        printf("\nTime to calc the tree: %f", tendcalc - tstartcalc);
        printf("\nTime to query the tree: %f", tendquery - tstartquery);
        printf(
            "\n[MODE: %d, N: %d] Total time: %f\n",
            MODE, N,
            (tendcalc - tstartcalc) + (tendquery - tstartquery)
        );

        return 0;
    }
    else if (MODE == 1)  // build and query kd-tree on CPU
    {
        tstartbuild = omp_get_wtime();
        kd_tree_cpu* tree = buildKdTreeCPU(dataset, N, DIM);
        struct kd_tree_node_gpu* gpu_nodes_array;
        allocate_gpu_memory(&(tree->root), &gpu_nodes_array, N);
        tendbuild = omp_get_wtime();

        printf("\n\nkd-tree:");
        print_tree(tree);
        printf("\n");

        tstartquery = omp_get_wtime();
        queryKdTreeCPU(&tree, result, dataset, epsilon, N, DIM);
        tendquery = omp_get_wtime();

        unsigned int totalWithinEpsilon = 0;
        for (unsigned int i = 0; i < N; i += 1)
        {
            totalWithinEpsilon += result[i];
        }

        printf("\nTotal number of points within epsilon: %u\n", totalWithinEpsilon);
        printf("\nTime to build the tree: %f", tendbuild - tstartbuild);
        printf("\nTime to query the tree: %f", tendquery - tstartquery);
        printf(
            "\n[MODE: %d, N: %d] Total time: %f\n",
            MODE, N,
            (tendbuild - tstartbuild) + (tendquery - tstartquery)
        );

        return 0;
    }

    double tstart=omp_get_wtime();

    //Allocate memory for the dataset
    float* dev_dataset;
    gpuErrchk(hipMalloc((float**)&dev_dataset, sizeof(float) * N * DIM));
    gpuErrchk(hipMemcpy(dev_dataset, dataset, sizeof(float) * N * DIM, hipMemcpyHostToDevice));

    //For part 1 that computes the distance matrix
    float* dev_distanceMatrix;
    gpuErrchk(hipMalloc((float**)&dev_distanceMatrix, sizeof(float) * N * N));

    //For part 2 for querying the distance matrix
    unsigned int* resultSet = (unsigned int*)calloc(N, sizeof(unsigned int));
    unsigned int* dev_resultSet;
    gpuErrchk(hipMalloc((unsigned int**)&dev_resultSet, sizeof(unsigned int) * N));
    gpuErrchk(hipMemcpy(dev_resultSet, resultSet, sizeof(unsigned int) * N, hipMemcpyHostToDevice));

    if (MODE == 2)  // brute force with GPU
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N * 1.0 / BLOCKDIM);
    }
    else if (MODE == 3)  // build tree on CPU and query on GPU
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N * 1.0 / BLOCKDIM);

        // build tree on CPU

        // move the tree onto the GPU
    }

    //Copy result set from the GPU
    gpuErrchk(hipMemcpy(resultSet, dev_resultSet, sizeof(unsigned int) * N, hipMemcpyDeviceToHost));

    //Compute the sum of the result set array
    unsigned int totalWithinEpsilon = 0;

    //Write code here
    for(int resultIndex = 0; resultIndex < N; resultIndex += 1)
    {
        totalWithinEpsilon += resultSet[resultIndex];
    }

    printf("\nTotal number of points within epsilon: %u", totalWithinEpsilon);

    double tend = omp_get_wtime();

    printf("\nTime to build the tree: %f\n", tendbuild - tstartbuild);
    printf("\nTime to query the tree: %f\n", tendquery - tstartquery);

    printf("\n[MODE: %d, N: %d] Total time: %f\n", MODE, N, tend - tstart);

    printf("\n\n");
    return 0;
}



void warmUpGPU()
{
    printf("\nWarming up GPU for time trialing...\n");
    hipDeviceSynchronize();
    return;
}



void checkParams(unsigned int N, unsigned int DIM)
{
    if (N <= 0 || DIM <= 0)
    {
        fprintf(stderr, "\n Invalid parameters: Error, N: %u, DIM: %u", N, DIM);
        fprintf(stderr, "\nReturning");
        exit(0);
    }
}



void importDataset(
        char* fname,
        unsigned int N,
        unsigned int DIM,
        float* dataset
) {
    FILE *fp = fopen(fname, "r");

    if (!fp)
    {
        fprintf(stderr, "Unable to open file\n");
        fprintf(stderr, "Error: dataset was not imported. Returning.");
        exit(0);
    }

    unsigned int bufferSize = DIM * 10;

    char buf[bufferSize];
    unsigned int rowCnt = 0;
    unsigned int colCnt = 0;
    while (fgets(buf, bufferSize, fp) && rowCnt < N)
    {
        colCnt = 0;

        char *field = strtok(buf, ",");
        double tmp;
        sscanf(field, "%lf", &tmp);

        dataset[rowCnt * DIM + colCnt] = tmp;


        while (field)
        {
            colCnt += 1;
            field = strtok(NULL, ",");

            if (field!=NULL)
            {
                double tmp;
                sscanf(field,"%lf",&tmp);
                dataset[rowCnt*DIM+colCnt]=tmp;
            }
        }

        rowCnt += 1;
    }

    fclose(fp);
}



// cpu
// brute force
void calcDistMatCPU(float* distanceMatrix, const float* dataset, const unsigned int N, const unsigned int DIM)
{
    for (unsigned int i = 0; i < N; i += 1)
    {
        for (unsigned int j = 0; j < N; j += 1)
        {
            float dist = 0.0;

            for (unsigned int d = 0; d < DIM; d += 1)
            {
                dist += (dataset[i * DIM + d] - dataset[j * DIM + d])
                    * (dataset[i * DIM + d] - dataset[j * DIM + d]);
            }

            distanceMatrix[i * N + j] = sqrt(dist);
        }
    }
}


void queryDistMat(unsigned int* result, const float* distanceMatrix, const float epsilon, const unsigned int N)
{
    for (unsigned int i = 0; i < N; i += 1)
    {
        for (unsigned int j = 0; j < N; j += 1)
        {
            if (distanceMatrix[i * N + j] <= epsilon)
            {
                result[i] += 1;
            }
        }
    }
}

// kd-tree
kd_tree_cpu* buildKdTreeCPU(const float* dataset, const unsigned int N, const unsigned int DIM)
{
    kd_tree_cpu* tree;

    init_kd_tree_cpu(&tree);

    for (unsigned int p = 0; p < N; p += 1)
    {
        kd_tree_node_cpu* node;
        float data[DIM];

        for (unsigned int d = 0; d < DIM; d += 1)
        {
            data[d] = dataset[p * DIM + d];
        }

        init_kd_tree_node_cpu(&node, data, DIM, 0);
        insert(&tree, &node);
    }

    return tree;
}


void queryKdTreeCPU(kd_tree_cpu** tree, unsigned int* result, const float* dataset, const float epsilon, const unsigned int N, const unsigned int DIM)
{
    float query[2];
    unsigned int count;
    for (unsigned int p = 0; p < N; p += 1)
    {
        count = 0;

        for (unsigned int d = 0; d < DIM; d += 1)
        {
            query[d] = dataset[p * DIM + d];
        }

        points_within_epsilon(tree, query, epsilon, &count);

        result[p] = count;
    }
}
=======
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <omp.h>
#include <iostream>
#include <complex.h>
#include <math.h>

#include "../include/kd_tree.h"


//Error checking GPU calls
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Mode 0 is CPU brute force - for checking tree results
// Mode 1 is CPU sequential implementation of kd-tree
// Mode 2 uses CPU to build the kd-tree and GPU to query
// Mode 3 ...
// ...
// #define MODE 0

//Define any constants here
#define BLOCKSIZE 128

using namespace std;


//function prototypes
void warmUpGPU();
void checkParams(unsigned int N, unsigned int DIM);

// cpu code
// brute force
void calcDistMatCPU(float* distanceMatrix, const float* dataset, const unsigned int N, const unsigned int DIM);
void calcQueryDistMat(unsigned int* result, const float* distanceMatrix, const float epsilon, const unsigned int N, const unsigned int DIM);

// gpu code

// handling data
void importDataset(
    char* fname,
    unsigned int N,
    unsigned int DIM,
    float* dataset
);



int main(int argc, char* argv[])
{
    printf("\nMODE: %d", MODE);
    warmUpGPU();

    char inputFname[500];
    unsigned int N = 0;
    unsigned int DIM = 0;
    float epsilon = 0;


    if (argc != 5)
    {
        fprintf(stderr, "Please provide the following on the command line: N (number of lines in the file), dimensionality (number of coordinates per point), epsilon, dataset filename.\n");
        exit(0);
    }

    sscanf(argv[1], "%d", &N);
    sscanf(argv[2], "%d", &DIM);
    sscanf(argv[3], "%f", &epsilon);
    strcpy(inputFname,argv[4]);

    checkParams(N, DIM);

    printf(
        "\nAllocating the following amount of memory for the dataset: %f GiB",
        (sizeof(float) * N * DIM) / (1024 * 1024 * 1024.0)
    );
    printf(
        "\nAllocating the following amount of memory for the distance matrix: %f GiB",
        (sizeof(float) * N * N) / (1024 * 1024 * 1024.0)
    );


    float* dataset = (float*)malloc(sizeof(float) * N * DIM);
    float* distanceMatrix = (float*)malloc(sizeof(float) * N * N);
    unsigned int* result = (unsigned int*)malloc(sizeof(unsigned int) * N);
    importDataset(inputFname, N, DIM, dataset);


    //CPU-only mode
    //It only computes the distance matrix but does not query the distance matrix
    if (MODE == 0)
    {
        double tstart = omp_get_wtime();

        calcDistMatCPU(distanceMatrix, dataset, N, DIM);
        calcQueryDistMat(result, distanceMatrix, epsilon, N, DIM);

        unsigned int totalWithinEpsilon = 0;
        for (unsigned int i = 0; i < N; i += 1)
        {
            totalWithinEpsilon += result[i];
        }

        double tend = omp_get_wtime();

        printf("\nTotal number of points within epsilon: %u", totalWithinEpsilon);
        printf("\n[MODE: %d, N: %d] Total time: %f", MODE, N, tend - tstart);

        return 0;
    }

    double tstart=omp_get_wtime();

    //Allocate memory for the dataset
    float* dev_dataset;
    gpuErrchk(hipMalloc((float**)&dev_dataset, sizeof(float) * N * DIM));
    gpuErrchk(hipMemcpy(dev_dataset, dataset, sizeof(float) * N * DIM, hipMemcpyHostToDevice));

    //For part 1 that computes the distance matrix
    float* dev_distanceMatrix;
    gpuErrchk(hipMalloc((float**)&dev_distanceMatrix, sizeof(float) * N * N));

    //For part 2 for querying the distance matrix
    unsigned int* resultSet = (unsigned int*)calloc(N, sizeof(unsigned int));
    unsigned int* dev_resultSet;
    gpuErrchk(hipMalloc((unsigned int**)&dev_resultSet, sizeof(unsigned int) * N));
    gpuErrchk(hipMemcpy(dev_resultSet, resultSet, sizeof(unsigned int) * N, hipMemcpyHostToDevice));

    //Baseline kernels
    if (MODE == 1)
    {
        unsigned int BLOCKDIM = BLOCKSIZE;
        unsigned int NBLOCKS = ceil(N*1.0/BLOCKDIM);

        // Call baseline kernel here
        // TODO: IMPLEMENT GPU IMPLEMENTATION
    }

    //Copy result set from the GPU
    gpuErrchk(hipMemcpy(resultSet, dev_resultSet, sizeof(unsigned int) * N, hipMemcpyDeviceToHost));

    //Compute the sum of the result set array
    unsigned int totalWithinEpsilon = 0;

    //Write code here
    for(int resultIndex = 0; resultIndex < N; resultIndex += 1)
    {
        totalWithinEpsilon += resultSet[resultIndex];
    }

    printf("\nTotal number of points within epsilon: %u", totalWithinEpsilon);

    double tend = omp_get_wtime();

    printf("\n[MODE: %d, N: %d] Total time: %f", MODE, N, tend-tstart);

    printf("\n\n");
    return 0;
}



void warmUpGPU()
{
    printf("\nWarming up GPU for time trialing...\n");
    hipDeviceSynchronize();
    return;
}



void checkParams(unsigned int N, unsigned int DIM)
{
    if (N <= 0 || DIM <= 0)
    {
        fprintf(stderr, "\n Invalid parameters: Error, N: %u, DIM: %u", N, DIM);
        fprintf(stderr, "\nReturning");
        exit(0);
    }
}



void importDataset(
        char* fname,
        unsigned int N,
        unsigned int DIM,
        float* dataset
) {
    FILE *fp = fopen(fname, "r");

    if (!fp)
    {
        fprintf(stderr, "Unable to open file\n");
        fprintf(stderr, "Error: dataset was not imported. Returning.");
        exit(0);
    }

    unsigned int bufferSize = DIM * 10;

    char buf[bufferSize];
    unsigned int rowCnt = 0;
    unsigned int colCnt = 0;
    while (fgets(buf, bufferSize, fp) && rowCnt < N)
    {
        colCnt = 0;

        char *field = strtok(buf, ",");
        double tmp;
        sscanf(field, "%lf", &tmp);
        
        dataset[rowCnt * DIM + colCnt] = tmp;

        
        while (field)
        {
            colCnt += 1;
            field = strtok(NULL, ",");

            if (field!=NULL)
            {
                double tmp;
                sscanf(field,"%lf",&tmp);
                dataset[rowCnt*DIM+colCnt]=tmp;
            }
        }

        rowCnt += 1;
    }

    fclose(fp);
}



// cpu
// brute force
void calcDistMatCPU(float* distanceMatrix, const float* dataset, const unsigned int N, const unsigned int DIM)
{
    for (unsigned int i = 0; i < N; i += 1)
    {
        for (unsigned int j = 0; j < N; j += 1)
        {
            float dist = 0.0;

            for (unsigned int d = 0; d < DIM; d += 1)
            {
                dist += (dataset[i * DIM + d] - dataset[i * DIM + d + 1])
                    * (dataset[i * DIM + d] - dataset[i * DIM + d + 1]);
            }

            distanceMatrix[i * N + j] = sqrt(dist);
        }
    }
}


void calcQueryDistMat(unsigned int* result, const float* distanceMatrix, const float epsilon, const unsigned int N, const unsigned int DIM)
{
    for (unsigned int i = 0; i < N; i += 1)
    {
        for (unsigned int j = 0; j < N; j += 1)
        {
            if (distanceMatrix[i * N + j] <= epsilon)
            {
                result[i] += 1;
            }
        }
    }
}

// kd-tree
>>>>>>> b48e6ae (Setup job script to run CPU brute force 3 times. Took out NAU ID in job script.)
